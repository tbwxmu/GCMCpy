#include "hip/hip_runtime.h"
/*
    © Copyright 2023 - University of Maryland, Baltimore   All Rights Reserved
        Mingtian Zhao, Alexander D. MacKerell Jr.
    E-mail:
        zhaomt@outerbanks.umaryland.edu
        alex@outerbanks.umaryland.edu
*/


// #include <unistd.h>
// #include <thrust/device_vector.h>
#include "gcmc.h"
#include "gcmc_move.h"

// #include <cstdio>
// #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != hipSuccess) 
//    {
//       fprintf(stderr,"GPU assert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }



extern "C"{

    __global__ void setup_rng_states(hiprandState *states, unsigned long long seed) {
        int global_threadIdx  = blockIdx.x * blockDim.x + threadIdx.x;
        hiprand_init(seed, global_threadIdx, 0, &states[global_threadIdx]);
    }
    // void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){}
    void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){
        
        InfoStruct *Ginfo;
        AtomArray *GfragmentInfo;
        residue *GresidueInfo; 
        Atom *GatomInfo;
        float *Ggrid;
        float *Gff;
        int *GmoveArray;

        hipMalloc(&Ginfo, sizeof(InfoStruct));
        hipMalloc(&GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum);
        hipMalloc(&GresidueInfo, sizeof(residue)*info->totalResNum);
        hipMalloc(&GatomInfo, sizeof(Atom)*info->totalAtomNum);
        hipMalloc(&Ggrid, sizeof(float)*info->totalGridNum * 3);
        hipMalloc(&Gff, sizeof(float)*info->ffXNum*info->ffYNum *2);
        // hipMalloc(&GmoveArray, sizeof(int)*info->mcsteps);

        // printf("hipMalloc done\n");

        // sleep(360);

        hipMemcpy(Ginfo, info, sizeof(InfoStruct), hipMemcpyHostToDevice);
        // printf("hipMemcpy Ginfo done\n");
        hipMemcpy(GfragmentInfo, fragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyHostToDevice);
        // printf("hipMemcpy GfragmentInfo done\n");
        hipMemcpy(GresidueInfo, residueInfo, sizeof(residue)*info->totalResNum, hipMemcpyHostToDevice) ;
        // printf("hipMemcpy GresidueInfo done\n");
        hipMemcpy(GatomInfo, atomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyHostToDevice);
        // printf("hipMemcpy GatomInfo done\n");
        hipMemcpy(Ggrid, grid, sizeof(float)*info->totalGridNum * 3, hipMemcpyHostToDevice);
        // printf("hipMemcpy Ggrid done\n");
        hipMemcpy(Gff, ff, sizeof(float)*info->ffXNum*info->ffYNum *2, hipMemcpyHostToDevice);
        // printf("hipMemcpy Gff done\n");
        // hipMemcpy(GmoveArray, moveArray, sizeof(int)*info->mcsteps, hipMemcpyHostToDevice);
        // printf("hipMemcpy GmoveArray done\n");


        // hipDeviceProp_t deviceProp;
        // int device;
        // hipGetDevice(&device);
        // hipGetDeviceProperties(&deviceProp, device);
        
        // int numberOfSMs = deviceProp.multiProcessorCount;
        
        // std::cout << "Number of SMs: " << numberOfSMs << std::endl;

        int maxConf = 0;
        for (int fragType = 0; fragType < info->fragTypeNum; fragType ++ ){

            if (fragmentInfo[fragType].confBias > maxConf){
                maxConf = fragmentInfo[fragType].confBias;
            }

        }

        AtomArray *GTempFrag;
        hipMalloc(&GTempFrag, sizeof(AtomArray)*maxConf);


        Atom *GTempInfo;
        hipMalloc(&GTempInfo, sizeof(Atom)*maxConf);

        Atom *TempInfo;
        TempInfo = (Atom *)malloc(sizeof(Atom)*maxConf);

        for (int i = 0;i < maxConf; i++){
            TempInfo[i].type = 0;
        }

        hipMemcpy(GTempInfo, TempInfo, sizeof(Atom)*maxConf, hipMemcpyHostToDevice);


        hiprandState *d_rng_states;
        
        hipMalloc((void **)&d_rng_states, maxConf * sizeof(hiprandState) * numThreadsPerBlock);


        srand(info->seed);

        setup_rng_states<<<maxConf, numThreadsPerBlock>>>(d_rng_states, info->seed);
        // printf("setup_rng_states done %u\n",info->seed);


        // std::cout << "maxConf: " << maxConf << std::endl;

        int step_threshold = info->mcsteps / 20; // Calculate 5% of total steps
        
        // // A simple progress indicator
        // for (int i=0;i < 10; i++ )
        // {
        //     printf("%d ", i);
        // }
        // printf("\n");

        for (int stepi = 0 ; stepi < info->mcsteps; ++stepi){
            // Start MC steps
            int moveFragType = moveArray[stepi] / 4;
            int moveMoveType = moveArray[stepi] % 4;
            int confBias = fragmentInfo[moveFragType].confBias;

            
            // // Print a dot every 10% of total steps
            // if (step_threshold !=0 && stepi % step_threshold == 0) {
            //     printf(".");
            //     fflush(stdout); // Ensure the dot is printed immediately
            // } 


            // perform move
            bool accepted = false;
            switch (moveMoveType)
            {
            case 0: // Insert
                accepted = move_add(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;

            case 1: // Del
                accepted = move_del(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;

            case 2: // Trn
                accepted = move_trn(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;

            case 3: // Rot
                accepted = move_rot(info, Ginfo,fragmentInfo, GfragmentInfo, GresidueInfo, GatomInfo, Ggrid, Gff, moveFragType, GTempFrag, TempInfo, GTempInfo, d_rng_states);
                break;
            }






        }
        printf("\n"); // Print a newline at the end of the simulation




        hipDeviceSynchronize();


        hipMemcpy(fragmentInfo, GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyDeviceToHost);
        hipMemcpy(residueInfo, GresidueInfo, sizeof(residue)*info->totalResNum, hipMemcpyDeviceToHost);
        hipMemcpy(atomInfo, GatomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyDeviceToHost);

        hipFree(Ginfo);
        hipFree(GfragmentInfo);
        hipFree(GresidueInfo);
        hipFree(GatomInfo);
        hipFree(Ggrid);
        hipFree(Gff);
        hipFree(GTempFrag);
        hipFree(GTempInfo);
        hipFree(d_rng_states);

        free(TempInfo);


        // hipFree(GmoveArray);
        
    }
}

// extern "C" {


//     void runGCMC_cuda(const InfoStruct *info, AtomArray *fragmentInfo, residue *residueInfo, Atom *atomInfo, const float *grid, const float *ff, const int *moveArray){


//         // InfoStruct *Ginfo;
//         // AtomArray *GfragmentInfo;
//         // residue *GresidueInfo; 
//         // Atom *GatomInfo;
//         // float *Ggrid;
//         // float *Gff;
//         // int *GmoveArray;


//         // hipMalloc(&Ginfo, sizeof(InfoStruct));

//         // hipMalloc((void**)&Ginfo, sizeof(InfoStruct));
//         // hipMalloc((void**)&GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum);
//         // hipMalloc((void**)&GresidueInfo, sizeof(residue)*info->totalResNum);
//         // hipMalloc((void**)&GatomInfo, sizeof(Atom)*info->totalAtomNum);
//         // hipMalloc((void**)&Ggrid, sizeof(float)*info->totalGridNum * 3);
//         // hipMalloc((void**)&Gff, sizeof(float)*info->ffXNum*info->ffYNum *2);
//         // hipMalloc((void**)&GmoveArray, sizeof(int)*info->mcsteps);

//         // hipMemcpy(Ginfo, info, sizeof(InfoStruct), hipMemcpyHostToDevice);
//         // hipMemcpy(GfragmentInfo, fragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyHostToDevice);
//         // hipMemcpy(GresidueInfo, residueInfo, sizeof(residue)*info->totalResNum, hipMemcpyHostToDevice);
//         // hipMemcpy(GatomInfo, atomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyHostToDevice);
//         // hipMemcpy(Ggrid, grid, sizeof(float)*info->totalGridNum * 3, hipMemcpyHostToDevice);
//         // hipMemcpy(Gff, ff, sizeof(float)*info->ffXNum*info->ffYNum *2, hipMemcpyHostToDevice);
//         // hipMemcpy(GmoveArray, moveArray, sizeof(int)*info->mcsteps, hipMemcpyHostToDevice);

        
//         // hipDeviceSynchronize();

//         // sleep(60);

//         // hipMemcpy(fragmentInfo, GfragmentInfo, sizeof(AtomArray)*info->fragTypeNum, hipMemcpyDeviceToHost);
//         // hipMemcpy(residueInfo, GresidueInfo, sizeof(residue)*info->totalResNum, hipMemcpyDeviceToHost);
//         // hipMemcpy(atomInfo, GatomInfo, sizeof(Atom)*info->totalAtomNum, hipMemcpyDeviceToHost);

//         // hipFree(Ginfo);
//         // hipFree(GfragmentInfo);
//         // hipFree(GresidueInfo);
//         // hipFree(GatomInfo);
//         // hipFree(Ggrid);
//         // hipFree(Gff);
//         // hipFree(GmoveArray);

//     }



// }

